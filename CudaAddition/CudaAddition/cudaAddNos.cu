#include "hip/hip_runtime.h"
/**
* Vector Addition - Simple addition using Cuda.
* Author - Malhar Bhatt
* Subject - High Performance Computing
*/

#include <iostream>
#include "headers.cuh"

/** Function Add - 
* Usage - Add 2 values
* Returns - Void
*/
__global__ void add( int num1, int num2, int *ans ) 
{
	*ans = num1 + num2;
}
int main( void ) 
{
	int numc;
	int *dev_numc;
	HANDLE_ERROR( hipMalloc( (void**)&dev_numc, sizeof(int) ) );
	add<<<1,1>>>( 10, 10, dev_numc ); // 1 thread and 1 kernel
	HANDLE_ERROR( hipMemcpy( &numc,dev_numc,sizeof(int),hipMemcpyDeviceToHost ) );
	printf( "10 + 10 = %d\n", numc );
	hipFree( dev_numc );
	system("pause");
	return 0;
}